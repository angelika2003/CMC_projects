#include "hip/hip_runtime.h"
#include "IL/il.h"
#include <stdlib.h>
#include <IL/devil_cpp_wrapper.hpp>
#include <iostream>
#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <string>
#include <chrono>

using namespace std;
#define SAFE_CALL(err) do { if (err != 0) { printf("ERROR [%s] in line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); exit(1); } } while(0)

__global__ void Kernel(int *arr, int size, ILubyte *Data_1, ILubyte*res, int h, int w) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
	for (int c = 0; c < 3; ++c) {
		int newValue = 0;
		int offset = size / 2;

		for (int i = 0; i < size; ++i) {
			int offsetY = y + i - offset;
			if (offsetY < 0) {
				offsetY = -offsetY;
			}
			if (offsetY >= h) {
				offsetY = 2 * h - offsetY - 1;
			}

			for (int j = 0; j < size; ++j) {
				int offsetX = x + j - offset;
				if (offsetX < 0) {
					offsetX = -offsetX;
				}
				if (offsetX >= w) {
					offsetX = 2 * w - offsetX - 1;
				}

				newValue += Data_1[(offsetY * w + offsetX) * 3 + c] * arr[i * size + j];
			}
		}
		// Ограничение значений от 0 до 255
		newValue = min(max(newValue, 0), 255);

		// Сохранение нового значения
		res[(y * w + x) * 3 + c] = static_cast<ILubyte>(newValue);
	}
}

int main(int argc, const char * argv[]) {
	int n = atoi(argv[1]);

	ILuint handle, w, h;

	/* First we initialize the library. */
	ilInit();

	/* We want all images to be loaded in a consistent manner */
	ilEnable(IL_ORIGIN_SET);

	/* In the next section, we load one image */
	ilGenImages(1, & handle);
	ilBindImage(handle);
	ILboolean loaded = ilLoadImage("pic.bmp");
	if (loaded == IL_FALSE)
	{
		printf("%s\n", iluErrorString(ilGetError()));
		return -1; /* error encountered during loading */
	}

	/* Let's spy on it a little bit */
	w = ilGetInteger(IL_IMAGE_WIDTH); // getting image width
	h = ilGetInteger(IL_IMAGE_HEIGHT); // and height
	printf("Our image resolution: %dx%d\n", w, h);

	/* how much memory will we need? */
	int memory_needed = w * h * 3 * sizeof(unsigned char);
	ILubyte * originalData = (ILubyte *)malloc(memory_needed);
	ilCopyPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, originalData);
	ILubyte * data = (ILubyte *)malloc(memory_needed);
	//ilCopyPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, data);
	ILubyte * data_gpu = (ILubyte *)malloc(memory_needed);
	//ilCopyPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, data_gpu);


	int *matrix = (int *)malloc(n * n * sizeof(int));
	for (int i = 0; i < n * n; i++) {
		matrix[i] = -1;
	}

	matrix[n * (n / 2) + n / 2] = n * n;

	int size = n;

	auto start = chrono::high_resolution_clock::now();
	for (int y = 0; y < h; ++y) {
		for (int x = 0; x < w; ++x) {
			for (int c = 0; c < 3; ++c) {
				int newValue = 0;
				int offset = size / 2;

				// Проход по окрестности пикселя с учетом матрицы фильтра
				for (int i = 0; i < size; ++i) {
					int offsetY = y + i - offset;
					if (offsetY < 0) {
						offsetY = -offsetY;
					}
					if (offsetY >= h) {
						offsetY = 2 * h - offsetY - 1;
					}

					for (int j = 0; j < size; ++j) {
						int offsetX = x + j - offset;
						if (offsetX < 0) {
							offsetX = -offsetX;
						}
						if (offsetX >= w) {
							offsetX = 2 * w - offsetX - 1;
						}

						newValue += originalData[(offsetY * w + offsetX) * 3 + c] * matrix[i * size + j];
					}
				}
				// Ограничение значений от 0 до 255
				newValue = min(max(newValue, 0), 255);

				// Сохранение нового значения
				data[(y * w + x) * 3 + c] = static_cast<ILubyte>(newValue);
			}
		}
	}

	auto end = chrono::high_resolution_clock::now();
	chrono::duration<double> duration = end - start;
	float time_cpu = duration.count();
	cout << "CPU : " << w*h / time_cpu / 1000000.0 << "mp/s" << endl;

	/* And maybe we want to save that all... */
	ilSetPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, data);
	/* and dump them to the disc... */
	ILboolean saved = ilSaveImage("result_cpu_5.bmp");
	if (saved == IL_FALSE)
	{
		printf("%s\n", iluErrorString(ilGetError()));
		return -1; /* error encountered during loading */
	}

	//memory on device:
	ILubyte * Data_1, *res;
	int* arr;
	SAFE_CALL(hipMalloc((void**)&Data_1, w * h * 3 * sizeof(unsigned char)));
	SAFE_CALL(hipMalloc((void**)&arr, n * n * sizeof(int)));
	SAFE_CALL(hipMalloc((void**)&res, w * h * 3 * sizeof(unsigned char)));

	//copy data from host to device
	SAFE_CALL(hipMemcpy(arr, matrix, n * n * sizeof(int) , hipMemcpyHostToDevice));

	SAFE_CALL(hipMemcpy(Data_1, originalData, w * h * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
	//SAFE_CALL(hipMemcpy(arr, matrix, n * n * sizeof(int) , hipMemcpyHostToDevice));

	hipEvent_t start_1, stop_1;
	float time_gpu;
	SAFE_CALL(hipEventCreate(&start_1));
	SAFE_CALL(hipEventCreate(&stop_1));
	SAFE_CALL(hipEventRecord(start_1, 0));


	dim3 blockDims(16, 16);
    dim3 gridDims((w + blockDims.x - 1) / blockDims.x, (h + blockDims.y - 1) / blockDims.y);
	Kernel <<< gridDims, blockDims >>>(arr, size, Data_1, res, h, w);
	SAFE_CALL(hipDeviceSynchronize());
	SAFE_CALL(hipEventRecord(stop_1, 0));
	SAFE_CALL(hipEventElapsedTime(&time_gpu, start_1, stop_1));
	time_gpu /= 1000;

	SAFE_CALL(hipEventDestroy(start_1));	
	SAFE_CALL(hipEventDestroy(stop_1));
	cout << "GPU : " << w*h / time_gpu / 1000000.0 << "mp/s" << endl;

	SAFE_CALL(hipMemcpy(data_gpu, res, w * h * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));



	/* And maybe we want to save that all... */
	ilSetPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, data_gpu);
	/* and dump them to the disc... */
	ILboolean saved_2 = ilSaveImage("result_gpu_5.bmp");
	if (saved_2 == IL_FALSE)
	{
		printf("%s\n", iluErrorString(ilGetError()));
		return -1; /* error encountered during loading */
	}

	// Finally, clean the mess! 
	ilDeleteImages(1, & handle);

	SAFE_CALL(hipFree(Data_1));
	SAFE_CALL(hipFree(arr));
	SAFE_CALL(hipFree(res));

	free(data); data = NULL;
	free(originalData);
	free(data_gpu);

	return 0;
}
