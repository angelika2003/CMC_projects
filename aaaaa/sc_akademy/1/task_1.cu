#include <iostream>
#include <random>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string>
#include <chrono>
#include <vector>

using namespace std;

__global__ void sum_kernel(double *A, double *B, double *C, int n, double a, double b) { 
	int treadLinearIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (treadLinearIdx < n) {
		double elemA = A[treadLinearIdx];
		double elemB = B[treadLinearIdx];
		C[treadLinearIdx] = elemA * a + elemB * b;
	}
}

#define SAFE_CALL(err) do { if (err != 0) { printf("ERROR [%s] in line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); exit(1); } } while(0)

int main(int argc, char **argv) {
	double a = stod(argv[1]);
	double b = stod(argv[2]);

	int n = stoi(argv[3]);
	int nb = n * sizeof(double);

	//only cpu code:////////////////
	//memory on host
	double* host_a, *host_b, *host_c, *host_c_gpu;
	SAFE_CALL(hipHostMalloc((void**)&host_a, nb, hipHostMallocDefault, hipHostMallocDefault));
	SAFE_CALL(hipHostMalloc((void**)&host_b, nb, hipHostMallocDefault, hipHostMallocDefault));
	SAFE_CALL(hipHostMalloc((void**)&host_c, nb, hipHostMallocDefault, hipHostMallocDefault));
	SAFE_CALL(hipHostMalloc((void**)&host_c_gpu, nb, hipHostMallocDefault, hipHostMallocDefault));

	random_device rd;
	mt19937 gen(rd());

	//fill host_a, host_b
	for (int i = 0; i < n; i++) {
		uniform_real_distribution<double> dis(0.0, 10000.0);

		host_a[i] = dis(gen);
		host_b[i] = dis(gen);
	}

	auto start = chrono::high_resolution_clock::now();
	for (int i = 0; i < n; i++) {
		host_c[i] = host_a[i] * a + host_b[i] * b;
	}

	auto end = chrono::high_resolution_clock::now();
	chrono::duration<double> duration = end - start;
	float time_cpu = duration.count();
	cout << "CPU time: " << time_cpu << "s" << endl;
	////////////////////////////////////////////////



	//memory on device:
	double* device_a, *device_b, *device_c;
	SAFE_CALL(hipMalloc((void**)&device_a, nb));
	SAFE_CALL(hipMalloc((void**)&device_b, nb));
	SAFE_CALL(hipMalloc((void**)&device_c, nb));

	hipEvent_t start_1, stop_1, start_2, stop_2;
	float time_gpu;
	float time_gpu_cp;

	SAFE_CALL(hipEventCreate(&start_2));
	SAFE_CALL(hipEventCreate(&stop_2));
	SAFE_CALL(hipEventRecord(start_2, 0));

	//copy data from host to device
	SAFE_CALL(hipMemcpy(device_a, host_a, nb, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(device_b, host_b, nb, hipMemcpyHostToDevice));


	SAFE_CALL(hipEventCreate(&start_1));
	SAFE_CALL(hipEventCreate(&stop_1));
	SAFE_CALL(hipEventRecord(start_1, 0));

	int threadsPerBlock = 512;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	sum_kernel <<< blocksPerGrid, threadsPerBlock>>>(device_a, device_b, device_c, n, a, b);

	SAFE_CALL(hipDeviceSynchronize());
	SAFE_CALL(hipEventRecord(stop_1, 0));
	SAFE_CALL(hipEventElapsedTime(&time_gpu, start_1, stop_1));
	time_gpu /= 1000;

	SAFE_CALL(hipMemcpy(host_c_gpu, device_c, nb, hipMemcpyDeviceToHost));
	SAFE_CALL(hipDeviceSynchronize());

	SAFE_CALL(hipEventRecord(stop_2, 0));
	SAFE_CALL(hipDeviceSynchronize());
	SAFE_CALL(hipEventElapsedTime(&time_gpu_cp, start_2, stop_2));
	time_gpu_cp /= 1000;

		
	SAFE_CALL(hipEventDestroy(start_1));
	SAFE_CALL(hipEventDestroy(stop_1));
	SAFE_CALL(hipEventDestroy(start_2));
	SAFE_CALL(hipEventDestroy(stop_2));

	cout << "GPU time: " << time_gpu << "s" << endl;
	cout << "GPU time with copy: " << time_gpu_cp << "s" << endl;

	cout << "acceleration: " << time_cpu / time_gpu << endl;
	cout << "acceleration with copy: " << time_cpu / time_gpu_cp << endl;




	int flag = 1;
	for (int i = 0; i < n; i++){
		if (host_c[i] - host_c_gpu[i] > 1e-5){
			flag = 0;
			break;
		}
	}

	if(flag){
		cout << "matrices are equivalent";
	} else {
		cout << "matrices are not equivalent";

	}

	SAFE_CALL(hipFree(device_a));
	SAFE_CALL(hipFree(device_b));
	SAFE_CALL(hipFree(device_c));

	SAFE_CALL(hipHostFree(host_a));
	SAFE_CALL(hipHostFree(host_b));
	SAFE_CALL(hipHostFree(host_c));
	SAFE_CALL(hipHostFree(host_c_gpu));


	return 0;
}
