#include "hip/hip_runtime.h"
#include "IL/il.h"
#include <stdlib.h>
#include <IL/devil_cpp_wrapper.hpp>
#include <iostream>
#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <string>
#include <chrono>
#include <fstream>
using namespace std;

bool compareImages(const std::string& imageFile1, const std::string& imageFile2) {
    // Открываем файлы с изображениями в двоичном режиме
    std::ifstream file1(imageFile1, std::binary);
    std::ifstream file2(imageFile2, std::binary);

    file1.seekg(0, std::ios::end);
    file2.seekg(0, std::ios::end);
    std::streamsize size1 = file1.tellg();
    std::streamsize size2 = file2.tellg();

    if (size1 != size2) {
        std::cout << "Размеры изображений не совпадают." << std::endl;
        return false;
    }

    // Возвращаем указатель чтения в начальное положение
    file1.seekg(0, std::beg);
    file2.seekg(0, std::beg);

    // Сравниваем пиксели изображений по байтам
    if (size1 > 0) {
        // Выделяем память для буферов
        char* buffer1 = new char[size1];
        char* buffer2 = new char[size1];

        // Читаем содержимое файлов в буферы
        file1.read(buffer1, size1);
        file2.read(buffer2, size1);

        // Сравниваем содержимое буферов
        bool result = (std::memcmp(buffer1, buffer2, size1) == 0);

        // Освобождаем выделенную память
        delete[] buffer1;
        delete[] buffer2;

        // Возвращаем результат сравнения
        return result;
    }

__device__ __host__ void func(int *arr, int size, ILubyte *Data_1, ILubyte*res, int h, int w, int n_str, int x, int y) {
	int newValue = 0;
	int offset = size / 2;

	for (int i = 0; i < size; ++i) {
		int offsetY = y + i - offset;
		if (offsetY < 0) {
			offsetY = -offsetY;
		}
		if (offsetY >= h) {
			offsetY = 2 * h - offsetY - 1;
		}

		for (int j = 0; j < size; ++j) {
			int offsetX = x + j - offset;
			if (offsetX < 0) {
				offsetX = -offsetX;
			}
			if (offsetX >= w) {
				offsetX = 2 * w - offsetX - 1;
			}

			newValue += Data_1[(offsetY * w + offsetX) * 3 + n_str] * arr[i * size + j];
		}
	}
	// Ограничение значений от 0 до 255
	newValue = min(max(newValue, 0), 255);

	// Сохранение нового значения
	res[(y * w + x) * 3 + n_str] = static_cast<ILubyte>(newValue);
	
}

#define SAFE_CALL(err) do { if (err != 0) { printf("ERROR [%s] in line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); exit(1); } } while(0)

__global__ void Kernel(int *arr, int size, ILubyte *Data_1, ILubyte*res, int h, int w, int n_str) {
	int x = blockIdx.x;
	int y = threadIdx.x;

	func(arr, size, Data_1, res, h, w, n_str, x, y);
}

int main(int argc, const char * argv[]) {
	int n = atoi(argv[1]);

	ILuint handle, w, h;

	/* First we initialize the library. */
	ilInit();

	/* We want all images to be loaded in a consistent manner */
	ilEnable(IL_ORIGIN_SET);

	/* In the next section, we load one image */
	ilGenImages(1, & handle);
	ilBindImage(handle);
	ILboolean loaded = ilLoadImage("pic.bmp");
	if (loaded == IL_FALSE)
	{
		printf("%s\n", iluErrorString(ilGetError()));
		return -1; /* error encountered during loading */
	}

	/* Let's spy on it a little bit */
	w = ilGetInteger(IL_IMAGE_WIDTH); // getting image width
	h = ilGetInteger(IL_IMAGE_HEIGHT); // and height
	printf("Our image resolution: %dx%d\n", w, h);

	/* how much memory will we need? */
	int memory_needed = w * h * 3 * sizeof(unsigned char);
	ILubyte * originalData = (ILubyte *)malloc(memory_needed);
	ilCopyPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, originalData);
	ILubyte * data = (ILubyte *)malloc(memory_needed);
	//ilCopyPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, data);
	ILubyte * data_gpu = (ILubyte *)malloc(memory_needed);
	//ilCopyPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, data_gpu);


	int *matrix = (int *)malloc(n * n * sizeof(int));
	for (int i = 0; i < n * n; i++) {
		matrix[i] = -1;
	}

	matrix[n * (n / 2) + n / 2] = n * n;

	int size = n;

	auto start = chrono::high_resolution_clock::now();
	for (int y = 0; y < h; ++y) {
		for (int x = 0; x < w; ++x) {
			for (int c = 0; c < 3; ++c) {
				func(matrix, size, originalData, data, h, w, c, x, y);
			}
		}
	}

	auto end = chrono::high_resolution_clock::now();
	chrono::duration<double> duration = end - start;
	float time_cpu = duration.count();
	cout << "CPU : " << w*h / time_cpu / 1000000.0 << "mp/s" << endl;

	/* And maybe we want to save that all... */
	ilSetPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, data);
	/* and dump them to the disc... */
	ILboolean saved = ilSaveImage("res_cpu_5.bmp");
	if (saved == IL_FALSE)
	{
		printf("%s\n", iluErrorString(ilGetError()));
		return -1; /* error encountered during loading */
	}

	//memory on device:
	ILubyte * Data_1, *res;
	int* arr;
	SAFE_CALL(hipMalloc((void**)&Data_1, w * h * 3 * sizeof(unsigned char)));
	SAFE_CALL(hipMalloc((void**)&arr, n * n * sizeof(int)));
	SAFE_CALL(hipMalloc((void**)&res, w * h * 3 * sizeof(unsigned char)));


	hipEvent_t start_1, stop_1;
	float time_gpu;
	SAFE_CALL(hipEventCreate(&start_1));
	SAFE_CALL(hipEventCreate(&stop_1));
	SAFE_CALL(hipEventRecord(start_1, 0));

////
	hipStream_t streams[3];
	for (int i = 0; i < 3; ++i) {
        SAFE_CALL(hipStreamCreate(&streams[i]));
    }
////

	//copy data from host to device
	SAFE_CALL(hipMemcpy(arr, matrix, n * n * sizeof(int) , hipMemcpyHostToDevice));

	SAFE_CALL(hipMemcpy(Data_1, originalData, w * h * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));


	Kernel <<< h, w, 0, streams[0] >>>(arr, size, Data_1, res, h, w, 0);
	Kernel <<< h, w, 0, streams[1] >>>(arr, size, Data_1, res, h, w, 1);
	Kernel <<< h, w, 0, streams[2] >>>(arr, size, Data_1, res, h, w, 2);

///////////

	SAFE_CALL(hipDeviceSynchronize());
	SAFE_CALL(hipEventRecord(stop_1, 0));
	SAFE_CALL(hipEventElapsedTime(&time_gpu, start_1, stop_1));
	time_gpu /= 1000;

	SAFE_CALL(hipEventDestroy(start_1));	
	SAFE_CALL(hipEventDestroy(stop_1));
	cout << "GPU : " << w*h / time_gpu / 1000000.0 << "mp/s" << endl;

	SAFE_CALL(hipMemcpy(data_gpu, res, w * h * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));



	/* And maybe we want to save that all... */
	ilSetPixels(0, 0, 0, w, h, 1, IL_RGB, IL_UNSIGNED_BYTE, data_gpu);
	/* and dump them to the disc... */
	ILboolean saved_2 = ilSaveImage("res_gpu_5.bmp");
	if (saved_2 == IL_FALSE)
	{
		printf("%s\n", iluErrorString(ilGetError()));
		return -1; /* error encountered during loading */
	}

//ident
	if (compareImages("res_cpu_5.bmp", "res_cpu_5.bmp")) {
        std::cout << "Изображения идентичны." << std::endl;
    } else {
        std::cout << "Изображения различаются." << std::endl;
    }	

	// Finally, clean the mess! 
	ilDeleteImages(1, & handle);

////
	for (int i = 0; i < 3; ++i) {
        SAFE_CALL(hipStreamDestroy(streams[i]));
    }
/////

	SAFE_CALL(hipFree(Data_1));
	SAFE_CALL(hipFree(arr));
	SAFE_CALL(hipFree(res));

	free(data); data = NULL;
	free(originalData);
	free(data_gpu);

	return 0;
}
